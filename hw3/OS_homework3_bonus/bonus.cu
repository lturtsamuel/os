
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <inttypes.h>

#define PAGESIZE 32

#define PHYSICAL_MEM_SIZE 32768

#define STORAGE_SIZE 131072

#define DATAFILE "./data.bin"
#define OUTFILE "./snapshot.bin"

#define MASK 32767
#define TIME_MAX 4294967295
#define MEMORY_SEGMENT 32768

#define __LOCK(); for(int j = 0; j < 4; j++) {if(threadIdx.x == j) {
#define __UNLOCK(); }__syncthreads(); }
#define __GET_BASE() j*MEMORY_SEGMENT
//C's macro is soooo bloody ugly comparing to ruby & lisp...

typedef unsigned char uchar;
typedef uint32_t u32;

//
__device__ __managed__ int PAGE_ENTRIES = PHYSICAL_MEM_SIZE/PAGESIZE;
//count the pagefault times
__device__ __managed__ int PAGEFAULT = 0;

//secondary memory
__device__ __managed__ uchar storage[STORAGE_SIZE];

//data input & output
__device__ __managed__ uchar result[STORAGE_SIZE];
__device__ __managed__ uchar input[STORAGE_SIZE];

//page table
extern __shared__ u32 pt[];

/******BLABLABLA~~****/
int load_binaryFile(const char *filename, uchar *a, int max_size) {
	FILE *fp = fopen(filename, "rb");
	int i = 0;
	while(!feof(fp) && i < max_size) {
		fread(a+i, sizeof(uchar), 1, fp);
		i++;
	}
	return i;
}

void write_binaryFIle(const char *filename, uchar *a, int size) {
	FILE *fp = fopen(filename, "wb+");
	fwrite(a, sizeof(uchar), size, fp);
}

__device__ u32 lru() {
	int offset = threadIdx.x*PAGE_ENTRIES/4;
	/****
	  實作queue來解決lru並無法解決效能瓶頸，因為最大的問題卡在find的O(n)
	  要改善find的效能，應實作binary search tree，but...
	 ***/
	u32 min = TIME_MAX;
	int victim_index = 0;
	for(int i = 0; i < PAGE_ENTRIES/4; i++) {
		if(pt[PAGE_ENTRIES+i+offset] == 0) return i+offset;
		else {
			if(pt[PAGE_ENTRIES+i+offset] < min) {
				min = pt[PAGE_ENTRIES+i+offset];
				victim_index = i;
			}
		}
	}
	return victim_index+offset;
}
__device__ int find(u32 p) {
	int offset = threadIdx.x*PAGE_ENTRIES/4;
	for(int i = 0; i < PAGE_ENTRIES/4; i++) {
		u32 cur_p= (pt[i+offset]>>15);
		if(cur_p == p) {
			if(pt[PAGE_ENTRIES+i+offset] == 0) return -1;
			else return i+offset;
		}
	}
	return -1;
}
__device__ u32 paging(uchar *data, u32 p, u32 offset) {
	if(pt[PAGE_ENTRIES*2] < TIME_MAX) pt[PAGE_ENTRIES*2]++;
	int p_index = find(p); //should only return the page that is of same id
	if(p_index == -1) {  //page fault!!
		PAGEFAULT++;
		u32 victim_index = lru(); //should only return the page that is of same id, since I can't see another thread's data[]
		u32 frame = pt[victim_index]&MASK;
		u32 victim_p = (pt[victim_index] >> 15);
		for(int i = 0; i < 32; i++) {
			storage[threadIdx.x*MEMORY_SEGMENT+victim_p*32+i] = data[frame+i];
			data[frame+i] = storage[threadIdx.x*MEMORY_SEGMENT+p*32+i];
		}
		pt[victim_index] = ((p<<15)|frame);
		pt[PAGE_ENTRIES+victim_index] = pt[PAGE_ENTRIES*2];
		return frame + offset;
	}
	else {
		pt[PAGE_ENTRIES+p_index] = pt[PAGE_ENTRIES*2];
		return (pt[p_index]&MASK) + offset;
	}
}
__device__ void init_pageTable(int pt_entries) {
	pt[PAGE_ENTRIES*2] = 0;
	for(int i = 0; i < PAGE_ENTRIES; i++) {
		pt[i] = (i*32)%(PAGE_ENTRIES/4);
		pt[PAGE_ENTRIES+i] = 0;
	}
}
/*********************/

__device__ uchar Gread(uchar *data, u32 addr) {
	u32 p = addr/PAGESIZE;
	u32 offset = addr%PAGESIZE;

	addr = paging(data, p, offset);
	return data[addr];
}

__device__ void Gwrite(uchar *data, u32 addr, uchar value) {
	u32 p = addr/PAGESIZE;
	u32 offset = addr%PAGESIZE;

	addr = paging(data, p, offset);
	data[addr] = value;
}

__device__ void snapshot(uchar *result, uchar *data, int offset, int input_size) {
	for(int i = 0; i < input_size; i++) {
		result[i] = Gread(data, i + offset);
		printf("id=%d, i=%d (%d, %d)\n", threadIdx.x, i, storage[i+MEMORY_SEGMENT*threadIdx.x], result[i]);
	}
}

__global__ void mykernel(int input_size) {
	__shared__ uchar data[32768];
	//get page table entries
	int pt_entries = PHYSICAL_MEM_SIZE/PAGESIZE;

	printf("my id = %d\n", threadIdx.x);
	//B4 1st Gwrite or Gread
	if(threadIdx.x == 0) init_pageTable(pt_entries);

	//####Gwrite/Gread code section start####
	__LOCK();
	for(int i = 0; i < input_size; i++) Gwrite(data, i, input[i+__GET_BASE()]);
	__UNLOCK();
	for(int i = input_size-1; i >= input_size-10; i--) {
		__LOCK();
		int value = Gread(data, i);
		__UNLOCK();
	}

	//the last line of Gwrite/Gread code section should be snapshot()
	__LOCK();
	snapshot(result+__GET_BASE(), data, 0, input_size);
	__UNLOCK();
	//####Gwrite/Gread code section end####
}

int main() {
	int input_size = load_binaryFile(DATAFILE, input, STORAGE_SIZE);

	hipSetDevice(3);
	mykernel<<<1, 4, 16384>>>(input_size/4);
	hipDeviceSynchronize();
	hipDeviceReset();

	printf("pagefault times = %d\n", PAGEFAULT);
	write_binaryFIle(OUTFILE, result, input_size);

	return 0;
}