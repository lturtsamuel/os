
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <inttypes.h>

#define PAGESIZE 32

#define PHYSICAL_MEM_SIZE 32768

#define STORAGE_SIZE 131072

#define DATAFILE "./data.bin"
#define OUTFILE "./snapshot.bin"

#define MASK 32767
#define TIME_MAX 4294967295
#define MEMORY_SEGMENT 32768

typedef unsigned char uchar;
typedef uint32_t u32;

//
__device__ __managed__ int PAGE_ENTRIES = PHYSICAL_MEM_SIZE/PAGESIZE;
//count the pagefault times
__device__ __managed__ int PAGEFAULT = 0;

//secondary memory
__device__ __managed__ uchar storage[STORAGE_SIZE];

//data input & output
__device__ __managed__ uchar result[STORAGE_SIZE];
__device__ __managed__ uchar input[STORAGE_SIZE];

//page table
extern __shared__ u32 pt[];

/******BLABLABLA~~****/
int load_binaryFile(const char *filename, uchar *a, int max_size) {
	FILE *fp = fopen(filename, "rb");
	int i = 0;
	while(!feof(fp) && i < max_size) {
		fread(a+i, sizeof(uchar), 1, fp);
		i++;
	}
	return i;
}

void write_binaryFIle(const char *filename, uchar *a, int size) {
	FILE *fp = fopen(filename, "wb+");
	fwrite(a, sizeof(uchar), size, fp);
}

__device__ u32 lru() {
	/****
	  實作queue來解決lru並無法解決效能瓶頸，因為最大的問題卡在find的O(n)
	  要改善find的效能，應實作binary search tree，but...
	 ***/
	u32 min = TIME_MAX;
	int victim_index = 0;
	for(int i = 0; i < PAGE_ENTRIES; i++) {
		if(pt[PAGE_ENTRIES+i] == 0) return i;
		else {
			if(pt[PAGE_ENTRIES+i] < min) {
				min = pt[PAGE_ENTRIES+i];
				victim_index = i;
			}
		}
	}
	return victim_index;
}
__device__ int find(u32 p) {
	for(int i = 0; i < PAGE_ENTRIES; i++) {
		u32 cur_p = (pt[i]>>15);
		if(cur_p == p) {
			if(pt[PAGE_ENTRIES+i] == 0) return -1;
			else return i;
		}
	}
	return -1;
}
__device__ u32 paging(uchar *data, u32 p, u32 offset) {
	if(pt[PAGE_ENTRIES*2] < TIME_MAX) pt[PAGE_ENTRIES*2]++;
	int p_index = find(p);
	if(p_index == -1) {  //page fault!!
		PAGEFAULT++;
		u32 victim_index = lru();
		u32 frame = pt[victim_index]&MASK;
		u32 victim_p = (pt[victim_index] >> 15);
		for(int i = 0; i < 32; i++) {
			storage[victim_p*32+i] = data[frame+i];
			data[frame+i] = storage[p*32+i];
		}
		pt[victim_index] = ((p<<15)|frame);
		pt[PAGE_ENTRIES+victim_index] = pt[PAGE_ENTRIES*2];
		return frame + offset;
	}
	else {
		pt[PAGE_ENTRIES+p_index] = pt[PAGE_ENTRIES*2];
		return (pt[p_index]&MASK) + offset;
	}
}
__device__ void init_pageTable(int pt_entries) {
	pt[PAGE_ENTRIES*2] = 0;
	for(int i = 0; i < PAGE_ENTRIES; i++) {
		pt[i] = i*32;
		pt[PAGE_ENTRIES+i] = 0;
	}
}
/*********************/

__device__ uchar Gread(uchar *data, u32 addr) {
	u32 p = addr/PAGESIZE;
	u32 offset = addr%PAGESIZE;

	addr = paging(data, p, offset);
	return data[addr];
}

__device__ void Gwrite(uchar *data, u32 addr, uchar value) {
	u32 p = addr/PAGESIZE;
	u32 offset = addr%PAGESIZE;

	addr = paging(data, p, offset);
	data[addr] = value;
}

__device__ void snapshot(uchar *result, uchar *data, int offset, int input_size) {
	for(int i = 0; i < input_size; i++) {
		result[i] = Gread(data, i + offset);
	}
}

__global__ void mykernel(int input_size) {
	__shared__ uchar data[PHYSICAL_MEM_SIZE];
	//get page table entries
	int pt_entries = PHYSICAL_MEM_SIZE/PAGESIZE;
	//B4 1st Gwrite or Gread
	init_pageTable(pt_entries);

	//####Gwrite/Gread code section start####
	for(int i = 0; i < input_size; i++) Gwrite(data, i, input[i]);
	for(int i = input_size-1; i >= input_size-10; i--) int value = Gread(data, i);

	//the last line of Gwrite/Gread code section should be snapshot()
	snapshot(result, data, 0, input_size);
	//####Gwrite/Gread code section end####
}

int main() {
	int input_size = load_binaryFile(DATAFILE, input, STORAGE_SIZE);

	hipSetDevice(3);
	mykernel<<<1, 1, 16384>>>(input_size);
	hipDeviceSynchronize();
	hipDeviceReset();

	printf("pagefault times = %d\n", PAGEFAULT);
	write_binaryFIle(OUTFILE, result, input_size);

	return 0;
}